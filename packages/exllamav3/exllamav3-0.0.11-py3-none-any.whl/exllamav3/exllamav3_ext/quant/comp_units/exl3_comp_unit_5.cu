#include <hip/hip_fp16.h>
#include <c10/cuda/CUDAGuard.h>
#include <ATen/cuda/HIPContext.h>
#include <hip/hip_cooperative_groups.h>
namespace cg = cooperative_groups;
#include "../../util.h"
#include "../../util.cuh"
#include "../../ptx.cuh"
#include "../exl3_gemm_kernel.cuh"
#include "exl3_comp_unit_5.cuh"

ALL_EXL3_KERNEL_INSTANCES(5)
