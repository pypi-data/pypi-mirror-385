#include <hip/hip_fp16.h>
#include <c10/cuda/CUDAGuard.h>
#include <ATen/cuda/HIPContext.h>
#include <hip/hip_cooperative_groups.h>
namespace cg = cooperative_groups;
#include "../util.h"
#include "../util.cuh"
#include "../ptx.cuh"
#include <tuple>
#include <mutex>
#include <map>
#include "exl3_kernel_map.cuh"
#include "exl3_devctx.cuh"
#include "comp_units/exl3_comp_unit_1.cuh"
#include "comp_units/exl3_comp_unit_2.cuh"
#include "comp_units/exl3_comp_unit_3.cuh"
#include "comp_units/exl3_comp_unit_4.cuh"
#include "comp_units/exl3_comp_unit_5.cuh"
#include "comp_units/exl3_comp_unit_6.cuh"
#include "comp_units/exl3_comp_unit_7.cuh"
#include "comp_units/exl3_comp_unit_8.cuh"

#include "exl3_kernel_map_samples.cuh"
std::map<uint64_t, TResult> _tuning_cache = {};

int select_gemm_shape(int cc, int size_m, int size_k, int size_n, int K, bool multi, int bszm_in, int bszm_out)
{
    bool mod_256 = (size_n % 256 == 0);
    bool mod_512 = (size_n % 512 == 0);

    size_k *= bszm_in;
    size_n *= bszm_out;

    switch(cc)
    {
        case CC_OLD:
        case CC_AMPERE:
            if (mod_256 && K <= 4)
            {
                if (size_n <= 2048 || size_k <= 2048) return 2;
                return 3;
            }
            if (mod_256 && size_n < 4096) return size_k > 8192 ? 3 : 2;
            if (mod_512 && (size_n * size_k) > (4096 * 4096) && K <= 6) return 4;
            if (mod_256) return 3;
            return 2;

        case CC_ADA:
            if (mod_256 && K <= 3)
            {
                if (size_k <= 2048 && !multi) return 2;
                if (size_n < 4096 && size_k <= 12288) return 2;
                return 3;
            }
            if (size_n <= 16384) return 2;
            if (mod_512 && size_n >= 32768) return 4;
            if (mod_256) return 3;
            return 2;

        // case CC_HOPPER:
        case CC_BLACKWELL:
            if ((K == 4 || K == 2) && !multi)
            {
                if (size_k <= 2048) return 1;
            }
            if (K >= 7)
            {
                if (mod_256 && size_n <= 8192) return size_k > 32768 ? 3 : 2;
                if (mod_512 && size_n > 32768) return 4;
                return 2;
            }
            if (mod_256 && size_n <= 4096) return size_k > 8192 && K >= 3 ? 3 : 2;
            if (mod_512 && size_n > 16384) return 4;
            if (mod_256) return 3;
            return 2;
    }
    return 0;
}

int exl3_gemm_num_kernel_shapes()
{
    return EXL3_GEMM_NUM_SHAPES;
}

int exl3_gemm_tilesize_k[] = {EXL3_GEMM_TILESIZE_K};
int exl3_gemm_tilesize_n[] = {EXL3_GEMM_TILESIZE_N};
int exl3_gemm_blockdim[] = {EXL3_GEMM_BLOCKDIM};

bool exl3_gemm_shape_compat(int shape_idx, int size_m, int size_k, int size_n, int K)
{
    int tilesize_k = exl3_gemm_tilesize_k[shape_idx];
    int tilesize_n = exl3_gemm_tilesize_n[shape_idx];
    return (size_k % tilesize_k == 0) && (size_n % tilesize_n == 0);
}

fp_exl3_gemm_kernel select_exl3_gemm_kernel
(
    int cc,
    int size_m,
    int size_k,
    int size_n,
    int K,
    bool c_fp32,
    int force_shape_idx,
    int* out_block_dim,
    int* out_shape_idx,
    int* num_sms,
    int cb
)
{
    int shape_idx = force_shape_idx <= 0 ? select_gemm_shape(cc, size_m, size_k, size_n, K, false, 1, 1) : force_shape_idx;

    TORCH_CHECK(shape_idx > 0, "exl3_gemm: no compatible kernel");
    if (out_shape_idx) *out_shape_idx = shape_idx;
    if (out_block_dim) *out_block_dim = exl3_gemm_blockdim[shape_idx];

    // Avoid empty blocks
    if (num_sms)
    {
        int tilesize_k = exl3_gemm_tilesize_k[shape_idx];
        int tilesize_n = exl3_gemm_tilesize_n[shape_idx];
        int max_slices = size_k / tilesize_k * size_n / tilesize_n;
        *num_sms = MAX(MIN(max_slices, *num_sms), 1);
    }

    int kernel_idx = shape_idx + (EXL3_GEMM_NUM_SHAPES + 1) * cb;

    if (c_fp32)
    {
        switch (K)
        {
            case 1: return tfp_exl3_gemm_kernel_fp32_b1[kernel_idx];
            case 2: return tfp_exl3_gemm_kernel_fp32_b2[kernel_idx];
            case 3: return tfp_exl3_gemm_kernel_fp32_b3[kernel_idx];
            case 4: return tfp_exl3_gemm_kernel_fp32_b4[kernel_idx];
            case 5: return tfp_exl3_gemm_kernel_fp32_b5[kernel_idx];
            case 6: return tfp_exl3_gemm_kernel_fp32_b6[kernel_idx];
            case 7: return tfp_exl3_gemm_kernel_fp32_b7[kernel_idx];
            case 8: return tfp_exl3_gemm_kernel_fp32_b8[kernel_idx];
            default: TORCH_CHECK(false, "No kernel for GEMM shape");
        }
    }
    else
    {
        switch (K)
        {
            case 1: return tfp_exl3_gemm_kernel_fp16_b1[kernel_idx];
            case 2: return tfp_exl3_gemm_kernel_fp16_b2[kernel_idx];
            case 3: return tfp_exl3_gemm_kernel_fp16_b3[kernel_idx];
            case 4: return tfp_exl3_gemm_kernel_fp16_b4[kernel_idx];
            case 5: return tfp_exl3_gemm_kernel_fp16_b5[kernel_idx];
            case 6: return tfp_exl3_gemm_kernel_fp16_b6[kernel_idx];
            case 7: return tfp_exl3_gemm_kernel_fp16_b7[kernel_idx];
            case 8: return tfp_exl3_gemm_kernel_fp16_b8[kernel_idx];
            default: TORCH_CHECK(false, "No kernel for GEMM shape");
        }
    }
}

fp_exl3_mgemm_kernel select_exl3_mgemm_kernel
(
    int cc,
    int size_m,
    int size_k,
    int size_n,
    int K,
    bool c_fp32,
    int force_shape_idx,
    int* out_block_dim,
    int* out_shape_idx,
    int* num_sms,
    int cb,
    int bszm_in,
    int bszm_out
)
{
    int shape_idx = force_shape_idx <= 0 ? select_gemm_shape(cc, size_m, size_k, size_n, K, true, bszm_in, bszm_out) : force_shape_idx;
    TORCH_CHECK(shape_idx > 0, "exl3_mgemm: no compatible kernel");
    if (out_shape_idx) *out_shape_idx = shape_idx;
    if (out_block_dim) *out_block_dim = exl3_gemm_blockdim[shape_idx];

    // Avoid empty blocks
    if (num_sms)
    {
        int tilesize_k = exl3_gemm_tilesize_k[shape_idx];
        int tilesize_n = exl3_gemm_tilesize_n[shape_idx];
        int max_slices = size_k / tilesize_k * size_n / tilesize_n / (*num_sms > 128 ? 20 : 24);
        *num_sms = MIN(max_slices, *num_sms);
    }

    int kernel_idx = shape_idx + (EXL3_GEMM_NUM_SHAPES + 1) * cb;

    if (c_fp32)
    {
        switch (K)
        {
            case 1: return tfp_exl3_mgemm_kernel_fp32_b1[kernel_idx];
            case 2: return tfp_exl3_mgemm_kernel_fp32_b2[kernel_idx];
            case 3: return tfp_exl3_mgemm_kernel_fp32_b3[kernel_idx];
            case 4: return tfp_exl3_mgemm_kernel_fp32_b4[kernel_idx];
            case 5: return tfp_exl3_mgemm_kernel_fp32_b5[kernel_idx];
            case 6: return tfp_exl3_mgemm_kernel_fp32_b6[kernel_idx];
            case 7: return tfp_exl3_mgemm_kernel_fp32_b7[kernel_idx];
            case 8: return tfp_exl3_mgemm_kernel_fp32_b8[kernel_idx];
            default: TORCH_CHECK(false, "No kernel for GEMM shape");
        }
    }
    else
    {
        switch (K)
        {
            case 1: return tfp_exl3_mgemm_kernel_fp16_b1[kernel_idx];
            case 2: return tfp_exl3_mgemm_kernel_fp16_b2[kernel_idx];
            case 3: return tfp_exl3_mgemm_kernel_fp16_b3[kernel_idx];
            case 4: return tfp_exl3_mgemm_kernel_fp16_b4[kernel_idx];
            case 5: return tfp_exl3_mgemm_kernel_fp16_b5[kernel_idx];
            case 6: return tfp_exl3_mgemm_kernel_fp16_b6[kernel_idx];
            case 7: return tfp_exl3_mgemm_kernel_fp16_b7[kernel_idx];
            case 8: return tfp_exl3_mgemm_kernel_fp16_b8[kernel_idx];
            default: TORCH_CHECK(false, "No kernel for GEMM shape");
        }
    }
}


fp_exl3_gemm_kernel get_gemm_kernel_ptr(int K, int shape_idx, bool c_fp32, int cb)
{
    int kernel_idx = shape_idx + (EXL3_GEMM_NUM_SHAPES + 1) * cb;

    if (c_fp32)
    {
        switch (K)
        {
            case 1: return tfp_exl3_gemm_kernel_fp32_b1[kernel_idx];
            case 2: return tfp_exl3_gemm_kernel_fp32_b2[kernel_idx];
            case 3: return tfp_exl3_gemm_kernel_fp32_b3[kernel_idx];
            case 4: return tfp_exl3_gemm_kernel_fp32_b4[kernel_idx];
            case 5: return tfp_exl3_gemm_kernel_fp32_b5[kernel_idx];
            case 6: return tfp_exl3_gemm_kernel_fp32_b6[kernel_idx];
            case 7: return tfp_exl3_gemm_kernel_fp32_b7[kernel_idx];
            case 8: return tfp_exl3_gemm_kernel_fp32_b8[kernel_idx];
            default: TORCH_CHECK(false, "No kernel for GEMM shape");
        }
    }
    else
    {
        switch (K)
        {
            case 1: return tfp_exl3_gemm_kernel_fp16_b1[kernel_idx];
            case 2: return tfp_exl3_gemm_kernel_fp16_b2[kernel_idx];
            case 3: return tfp_exl3_gemm_kernel_fp16_b3[kernel_idx];
            case 4: return tfp_exl3_gemm_kernel_fp16_b4[kernel_idx];
            case 5: return tfp_exl3_gemm_kernel_fp16_b5[kernel_idx];
            case 6: return tfp_exl3_gemm_kernel_fp16_b6[kernel_idx];
            case 7: return tfp_exl3_gemm_kernel_fp16_b7[kernel_idx];
            case 8: return tfp_exl3_gemm_kernel_fp16_b8[kernel_idx];
            default: TORCH_CHECK(false, "No kernel for GEMM shape");
        }
    }
}


fp_exl3_mgemm_kernel get_mgemm_kernel_ptr(int K, int shape_idx, bool c_fp32, int cb)
{
    int kernel_idx = shape_idx + (EXL3_GEMM_NUM_SHAPES + 1) * cb;

    if (c_fp32)
    {
        switch (K)
        {
            case 1: return tfp_exl3_mgemm_kernel_fp32_b1[kernel_idx];
            case 2: return tfp_exl3_mgemm_kernel_fp32_b2[kernel_idx];
            case 3: return tfp_exl3_mgemm_kernel_fp32_b3[kernel_idx];
            case 4: return tfp_exl3_mgemm_kernel_fp32_b4[kernel_idx];
            case 5: return tfp_exl3_mgemm_kernel_fp32_b5[kernel_idx];
            case 6: return tfp_exl3_mgemm_kernel_fp32_b6[kernel_idx];
            case 7: return tfp_exl3_mgemm_kernel_fp32_b7[kernel_idx];
            case 8: return tfp_exl3_mgemm_kernel_fp32_b8[kernel_idx];
            default: TORCH_CHECK(false, "No kernel for GEMM shape");
        }
    }
    else
    {
        switch (K)
        {
            case 1: return tfp_exl3_mgemm_kernel_fp16_b1[kernel_idx];
            case 2: return tfp_exl3_mgemm_kernel_fp16_b2[kernel_idx];
            case 3: return tfp_exl3_mgemm_kernel_fp16_b3[kernel_idx];
            case 4: return tfp_exl3_mgemm_kernel_fp16_b4[kernel_idx];
            case 5: return tfp_exl3_mgemm_kernel_fp16_b5[kernel_idx];
            case 6: return tfp_exl3_mgemm_kernel_fp16_b6[kernel_idx];
            case 7: return tfp_exl3_mgemm_kernel_fp16_b7[kernel_idx];
            case 8: return tfp_exl3_mgemm_kernel_fp16_b8[kernel_idx];
            default: TORCH_CHECK(false, "No kernel for GEMM shape");
        }
    }
}


TResult f_tr;

TResult* select_exl3_gemm_mgemm_kernel_new
(
    int cc,
    int size_m,
    int size_k,
    int size_n,
    int K,
    bool c_fp32,
    int force_shape_idx,
    int force_num_sms,
    int cb
)
{
    // Force parameters for tuning/benchmarking
    if (force_shape_idx > 0)
    {
        TORCH_CHECK(force_num_sms, "Must supply force_shape_idx and force_num_sms together");
        f_tr.kernel = get_gemm_kernel_ptr(K, force_shape_idx, c_fp32, cb);
        f_tr.mkernel = get_mgemm_kernel_ptr(K, force_shape_idx, c_fp32, cb);
        f_tr.shape_idx = force_shape_idx;
        f_tr.num_sms = force_num_sms;
        f_tr.block_dim = exl3_gemm_blockdim[force_shape_idx];
        return &f_tr;
    };
    TORCH_CHECK(!force_num_sms, "Must supply force_shape_idx and force_num_sms together.");

    // Cache parameters
    uint64_t key = (((uint64_t) size_k) << 40) |
                   (((uint64_t) size_n) << 16) |
                   (((uint64_t) cc)     <<  8) |
                   (((uint64_t) K)      <<  4) |
                   (c_fp32 ? 0x01ull : 0x00ull);

    auto lookup = _tuning_cache.find(key);
    if (lookup == _tuning_cache.end())
    {
        // Find closest kernel in map
        bool mod512 = (size_n % 512 == 0);
        bool mod256 = (size_n % 256 == 0);
        bool mod128 = (size_n % 128 == 0);
        TORCH_CHECK(mod128, "size_n must be a multiple of 128");
        TSample* cand = mod512 ? samples_512 : (mod256 ? samples_256 : samples_128);
        TSample* best = nullptr;
        int64_t best_dist = 1ll<<62;

        for (; cand->K; cand++)
        {
            if (cand->K != K) continue;
            if (cand->cc != cc) continue;

            int64_t distk = (int64_t) (size_k - cand->k);
            int64_t distn = (int64_t) (size_n - cand->n);
            int64_t dist = distk * distk + distn * distn;
            if (dist < best_dist) { best_dist = dist; best = cand; }
        }
        TORCH_CHECK(best, "Failed to find valid kernel for shape");

        // Avoid empty blocks
        int tilesize_k = exl3_gemm_tilesize_k[best->shape_idx];
        int tilesize_n = exl3_gemm_tilesize_n[best->shape_idx];
        int max_slices = size_k / tilesize_k * size_n / tilesize_n;
        int num_sms = MAX(MIN(max_slices, best->num_sms), 1);

        // Results
        TResult tr = {
            get_gemm_kernel_ptr(K, best->shape_idx, c_fp32, cb),
            get_mgemm_kernel_ptr(K, best->shape_idx, c_fp32, cb),
            best->shape_idx,
            num_sms,
            exl3_gemm_blockdim[best->shape_idx]
        };

        _tuning_cache[key] = tr;
    }

    lookup = _tuning_cache.find(key);
    return &(lookup->second);
}