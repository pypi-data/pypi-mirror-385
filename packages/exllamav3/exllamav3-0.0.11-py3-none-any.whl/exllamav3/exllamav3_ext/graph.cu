#include <Python.h>
#include "graph.cuh"
#include <c10/cuda/CUDAGuard.h>
#include <ATen/cuda/HIPContext.h>
//#include <torch/extension.h>
#include "util.h"
#include "util.cuh"

Graph::Graph()
{
    ready = false;
    graph = NULL;
    graph_exec = NULL;
}

Graph::~Graph()
{
    if (graph) hipGraphDestroy(graph);
    if (graph_exec) hipGraphExecDestroy(graph_exec);
}

hipStream_t Graph::capture_begin()
{
    // Make sure nothing is pending
    hipDeviceSynchronize();

    // Create capture stream
    cuda_check(hipStreamCreateWithFlags(&capture_stream, hipStreamNonBlocking));

    // Begin capture
    cuda_check(hipStreamBeginCapture(capture_stream, hipStreamCaptureModeThreadLocal));
    return capture_stream;
}

void Graph::capture_end()
{
    // End capture
    cuda_check(hipStreamEndCapture(capture_stream, &graph));
    cuda_check(hipGraphInstantiate(&graph_exec, graph, nullptr, nullptr, 0));
    //inspect_graph();

    // Get graph nodes
    size_t num_nodes;
    hipGraphGetNodes(graph, nullptr, &num_nodes);
    nodes.resize(num_nodes);
    hipGraphGetNodes(graph, nodes.data(), &num_nodes);

    // Store copies of all node param structures
    node_params.resize(num_nodes);
    node_needs_update.resize(num_nodes);
    for (int i = 0; i < num_nodes; ++i)
        node_needs_update[i] = false;

    int n = 0;
    int c = 0;
    while (true)
    {
        hipGraphNodeType t{};
        hipGraphNodeGetType(nodes[n], &t);

        // Node type: kernel
        if (t == hipGraphNodeTypeKernel)
        {
            hipGraphKernelNodeGetParams(nodes[n], &node_params[n]);
//            DBGX(node_params[n].func);

            for(; c < graph_sites.size(); c++)
            {
                void* func = std::get<0>(graph_sites[c]);
//                DBGX(func);

                if (func != node_params[n].func) break;

                int param_id     = std::get<1>(graph_sites[c]);
                int param_offset = std::get<2>(graph_sites[c]);

                graph_node_sites.push_back(std::tuple<int, int, int>(n, param_id, param_offset));
                if (param_id == GP_end) { c++; break; }

//                DBGI2(param_id, param_offset);
            }
        }

        n++;
        if (c == graph_sites.size()) break;
        if (n == num_nodes) TORCH_CHECK(false, "Graph recording failed");
    };

    // Destroy capture stream
    cuda_check(hipStreamDestroy(capture_stream));

    // Graph is ready
    ready = true;
}

void Graph::record_param(void* kernel, int param_id, int param_offset)
{
    graph_sites.push_back(std::tuple<void*, int, int>(kernel, param_id, param_offset));
}

void Graph::launch(std::vector<PPTR> params, hipStream_t stream)
{
    int p = 0;
    int n = 0;
    while (true)
    {
        if (std::get<1>(graph_node_sites[n]) == std::get<0>(params[p]))
        {
            if (std::get<0>(params[p]) != GP_end)
            {
                void* new_value  = std::get<1>(params[p]);
                int node_idx     = std::get<0>(graph_node_sites[n]);
                int param_offset = std::get<2>(graph_node_sites[n]);

//                DBGI3(p, node_idx, param_offset);

                void** p_old_value = (void**) node_params[node_idx].kernelParams[param_offset];
                if (*p_old_value != new_value)
                {
                    *p_old_value = new_value;
                    node_needs_update[node_idx] = true;
                }
            }
            else
            {
//                DBGI(p);
            }
            p++;
        }

        n++;
        if (p == params.size()) break;
        if (n == graph_node_sites.size()) TORCH_CHECK(false, "Graph update failed");
    }

    for (int n = 0; n < nodes.size(); ++n)
    {
//        DBGI(n);
        if (!node_needs_update[n]) continue;
//        printf("update\n");
        hipGraphExecKernelNodeSetParams(graph_exec, nodes[n], &node_params[n]);
        node_needs_update[n] = false;
    }

    hipGraphLaunch(graph_exec, stream);
}

void Graph::inspect_graph()
{
    // Get the number of nodes in the graph
    size_t numNodes;
    hipGraphGetNodes(graph, nullptr, &numNodes);

    // Get the nodes in the graph
    std::vector<hipGraphNode_t> nodes(numNodes);
    hipGraphGetNodes(graph, nodes.data(), &numNodes);
    DBGI(nodes.size());

    // Inspect each node
    for (size_t i = 0; i < numNodes; ++i)
    {
        hipGraphNodeType nodeType;
        hipGraphNodeGetType(nodes[i], &nodeType);

        if (nodeType == hipGraphNodeTypeKernel)
        {
            hipKernelNodeParams nodeParams;
            hipGraphKernelNodeGetParams(nodes[i], &nodeParams);
            std::cout << "Kernel node " << i << ":" << std::endl;
            std::cout << "  Function pointer: " << nodeParams.func << std::endl;
            std::cout << "  Grid dimensions: (" << nodeParams.gridDim.x << ", " << nodeParams.gridDim.y << ", " << nodeParams.gridDim.z << ")" << std::endl;
            std::cout << "  Block dimensions: (" << nodeParams.blockDim.x << ", " << nodeParams.blockDim.y << ", " << nodeParams.blockDim.z << ")" << std::endl;
            std::cout << "  Shared memory: " << nodeParams.sharedMemBytes << " bytes" << std::endl;

        } else {
            std::cout << "Node " << i << " is not a kernel node." << std::endl;
        }
    }
}

