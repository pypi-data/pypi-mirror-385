/* -------------------------------------------------------------------------- */
// Half-failed attempt to find monoliths with cuda.
//
// I'm somewhat sure the lack of memory locality between different seeds and
// the large-ish fractal perlin structs for the tiny cores killed performance.
//
// Maybe it's possible to fix it, or perhaps certain computations are just
// better suited for the CPU even if embarrassingly parallel. This was my first
// time toying with cuda, chances are I did something wrong, PRs are welcome
// to improve this code for the endeavor to find the largest monoliths :^)
//
// Compiling and running:
// - Install CUDA from your package manager (nvcc), have it on path, good luck
//
// Either manually:
//   - Run: `meson setup --buildtype release ./build --reconfigure`
//   - Run: `ninja -C ./build`
//   - Run: `./build/monolithium`
//
// Or install [uv](https://docs.astral.sh/uv/):
//   - Run: `uvx run cudalith`
/* -------------------------------------------------------------------------- */

#include <cstdint>
#include <cstdio>

#include <hip/hip_runtime.h>

// Having trouble with names..
#define Gpu __device__
#define Cpu __host__

// Read Cargo.toml for more info!
#define SKIP_REJECTION 1
#define SKIP_TABLE 1

/* -------------------------------------------------------------------------- */
// Utility functions

Gpu inline float fade(float t) {
    return t * t * t * (t * (t * 6.0 - 15.0) + 10.0);
}

Gpu inline float lerp(float t, float a, float b) {
    // return a + t * (b - a);
    return fma(t, b - a, a);
}

Gpu const float GRAD_LOOKUP[16][3] = {
    { 1.0f,  1.0f,  0.0f}, //  0:  x + y
    {-1.0f,  1.0f,  0.0f}, //  1: -x + y
    { 1.0f, -1.0f,  0.0f}, //  2:  x - y
    {-1.0f, -1.0f,  0.0f}, //  3: -x - y
    { 1.0f,  0.0f,  1.0f}, //  4:  x + z
    {-1.0f,  0.0f,  1.0f}, //  5: -x + z
    { 1.0f,  0.0f, -1.0f}, //  6:  x - z
    {-1.0f,  0.0f, -1.0f}, //  7: -x - z
    { 0.0f,  1.0f,  1.0f}, //  8:  y + z
    { 0.0f, -1.0f,  1.0f}, //  9: -y + z
    { 0.0f,  1.0f, -1.0f}, // 10:  y - z
    { 0.0f, -1.0f, -1.0f}, // 11: -y - z
    { 1.0f,  1.0f,  0.0f}, // 12:  y + x
    { 0.0f, -1.0f,  1.0f}, // 13: -y + z
    {-1.0f,  1.0f,  0.0f}, // 14:  y - x
    { 0.0f, -1.0f, -1.0f}, // 15: -y - z
};

Gpu inline float grad(uint8_t hash, float x, float y, float z) {
    const float* weights = GRAD_LOOKUP[hash & 0x0F];
    return fma(weights[0], x, fma(weights[1], y, weights[2] * z));
}

/* -------------------------------------------------------------------------- */
// Java RNG implementation

constexpr double F64_DIV = (1ULL << 53);
constexpr uint64_t M = (1LL << 48) - 1;
constexpr uint64_t A = 0x5DEECE66DLL;
constexpr uint64_t C = 11LL;

struct JavaRNG {
    uint64_t state;

    Gpu inline JavaRNG(uint64_t seed) {
        this->state = ((uint64_t) seed ^ A) & M;
    }

    // Roll the state, same effect as ignoring a `.next()` call
    Gpu inline void step() {
        this->state = (this->state * A + C) & M;
    }

    // Rolls the state and returns N low bits
    Gpu inline int next(uint8_t bits) {
        this->step();
        return (int) (this->state >> (48 - bits));
    }

    Gpu inline int next_i32_bound(int max) {
        if (__popc(max) == 1) {
            return (int)(((int64_t) max * (int64_t) this->next(31)) >> 31);
        }

        int next = this->next(31);
        int take = next % max;

        #if SKIP_REJECTION
            // Nothing out of the ordinary, I mean
        #else
            while (next - take + max - 1 < 0) {
                next = this->next(31);
                take = next % max;
            }
        #endif

        return take;
    }

    Gpu inline double next_f64() {
        int64_t high = ((int64_t) this->next(26)) << 27;
        int64_t low  =  (int64_t) this->next(27);
        return (double)(high | low) / F64_DIV;
    }

    // Lossy step the state as if a perlin noise was created
    Gpu inline void step_perlin() {
        // SKIP_TABLE[256 + 3*2]
        this->state *= 253119540505593LL;
        this->state += 184089911826014LL;
        this->state &= M;
    }

    // Lossy step the state as if 48 perlin noises were created
    Gpu inline void step_48_perlin() {
        // SKIP_TABLE[48*(256 + 3*2)]
        this->state *= 249870891710593LL;
        this->state += 44331453843488LL;
        this->state &= M;
    }
};

/* -------------------------------------------------------------------------- */

struct PerlinNoise {
    uint8_t map[256];
    float xoff;
    float yoff;
    float zoff;

    Gpu void init(JavaRNG* rng) {
        this->xoff = (float) (rng->next_f64() * 256.0);
        this->yoff = (float) (rng->next_f64() * 256.0);
        this->zoff = (float) (rng->next_f64() * 256.0);

        // Start a new 'arange' array
        for (int i=0; i<256; i++) {
            this->map[i] = i;
        }

        uint8_t temp;

        // Fixme: The memory swap is one of the most expensive operations in
        //   the whole code, is there a better way to do this?
        for (int a=0; a<256; a++) {
            int b = a + rng->next_i32_bound(256 - a);
            temp = this->map[a];
            this->map[a] = this->map[b];
            this->map[b] = temp;
        }
    }

    // Lazy and DRY to write bitwise everywhere
    Gpu inline uint8_t get_map(uint8_t index) {
        return this->map[index & 0xFF];
    }

    // Fixme: Second-most expensive method, can we avoid random memory access?
    // Sample the noise at a given coordinate
    // - Note: For monoliths, y is often 0.0
    Gpu float sample(float x, float y, float z) {
        x += this->xoff;
        y += this->yoff;
        z += this->zoff;

        // Convert to grid coordinates (512 length)
        int xi = __float2int_rd(x) & 0xFF;
        int yi = __float2int_rd(y) & 0xFF;
        int zi = __float2int_rd(z) & 0xFF;

        // Get the fractional parts
        float xf = x - floor(x);
        float yf = y - floor(y);
        float zf = z - floor(z);

        // Smoothstep-like factors
        float u = fade(xf);
        float v = fade(yf);
        float w = fade(zf);

        // Get the hash values for the corners
        uint8_t a  = this->get_map(xi + 0 + 0);
        uint8_t aa = this->get_map(yi + a + 0);
        uint8_t ab = this->get_map(yi + a + 1);
        uint8_t b  = this->get_map(xi + 0 + 1);
        uint8_t ba = this->get_map(yi + b + 0);
        uint8_t bb = this->get_map(yi + b + 1);

        return lerp(w,
            lerp(v,
                lerp(u, grad(this->get_map(aa + zi), xf, yf, zf),
                        grad(this->get_map(ba + zi), xf - 1.0, yf, zf)),
                lerp(u, grad(this->get_map(ab + zi), xf, yf - 1.0, zf),
                        grad(this->get_map(bb + zi), xf - 1.0, yf - 1.0, zf))
            ),
            lerp(v,
                lerp(u, grad(this->get_map(aa + zi + 1), xf, yf, zf - 1.0),
                        grad(this->get_map(ba + zi + 1), xf - 1.0, yf, zf - 1.0)),
                lerp(u, grad(this->get_map(ab + zi + 1), xf, yf - 1.0, zf - 1.0),
                        grad(this->get_map(bb + zi + 1), xf - 1.0, yf - 1.0, zf - 1.0))
            )
        );
    }

    // Roll the generator state that would have created a PerlinNoise
    // - Fast way around without as many memory operations
    Gpu static void discard(JavaRNG* rng, int count) {
        for (int i=0; i<count; i++) {

            // Coordinates f64 offsets
            for (int j=0; j<3; j++) {
                rng->next_f64();
            }

            // Permutations swapping
            for (int max=256; max>=1; max--) {
                rng->next_i32_bound(max);
            }
        }
    }
};

/* -------------------------------------------------------------------------- */

template<int OCTAVES> struct FractalPerlin {
    PerlinNoise noise[OCTAVES];

    Gpu void init(JavaRNG* rng) {
        for (uint8_t i=0; i<OCTAVES; i++) {
            this->noise[i].init(rng);
        }
    }

    Gpu inline float octave_scale(uint8_t index) {
        return (1 << index);
    }

    Gpu float sample(float x, float y, float z) {
        float sum = 0.0f;
        for (uint8_t i=0; i<OCTAVES; i++) {
            float s = this->octave_scale(i);
            sum += this->noise[i].sample(x/s, y/s, z/s) * s;
        }
        return sum;
    }
};

/* -------------------------------------------------------------------------- */

struct World {
    FractalPerlin<10> hill;
    FractalPerlin<16> depth;

    Gpu void init(uint64_t seed) {
        JavaRNG rng(seed);

        // Skip 48 generators priorly used elsewhere
        #if SKIP_TABLE
            rng.step_48_perlin();
        #else
            PerlinNoise::discard(&rng, 48);
        #endif

        this->hill.init(&rng);
        this->depth.init(&rng);
    }

    // Check if a given coordinate is part of a monolith
    Gpu bool is_monolith(int x, int z) {
        return fabs(this->depth.sample(
            (float) (x/4) * 100.0, 0.0,
            (float) (z/4) * 100.0
        )) > 8000.0f
        &&
        this->hill.sample(
            (float) (x/4) * 1.0, 0.0,
            (float) (z/4) * 1.0
        ) < -512.0f;
    }

    Gpu bool around_spawn(int radius, int step) {
        for (int x=-radius; x<=radius; x+=step) {
            for (int z=-radius; z<=radius; z+=step) {
                if (this->is_monolith(x, z)) {
                    return true;
                }
            }
        }
        return false;
    }
};

/* -------------------------------------------------------------------------- */

__global__ void get_monoliths_world_per_block(
    int start, int seeds,
    float* results
) {
    int idx  = threadIdx.x;
    int dim  = blockDim.x;
    int blk  = blockIdx.x;
    int seed = start + blk;

    __shared__ World world;

    if (threadIdx.x == 0) {
        world.init(seed);

        if (blockIdx.x % 10000 == 0)
            printf("Block %d seed %d\n", blk, seed);

        if (!world.around_spawn(200, 100))
            return;
    }

    const int side = 4096;
    const int step = 32;
    float area = 0;

    // Each thread sums its strip
    for (int x=-side+idx; x<=side; x+=step*dim) {
        for (int z=-side; z<=side; z+=step) {
            area += (int) world.is_monolith(x, z) * (step*step);
        }
    }

    atomicAdd(&results[blk], area);
}

__global__ void get_monoliths_world_per_thread(
    int start, int seeds,
    float* results
) {
    int tdx  = threadIdx.x;
    int dim  = blockDim.x;
    int blk  = blockIdx.x;
    int tid  = (blk * dim) + tdx;
    int seed = start + tid;

    World world;
    world.init(seed);

    if (tid % 100000 == 0)
        printf("Block %d seed %d\n", blk, seed);

    if (!world.around_spawn(200, 200))
        return;

    const int side = 256;
    const int step = 4;

    // Todo: BFS-like approach from Rust
    for (int x=-side; x<=side; x+=step) {
        for (int z=-side; z<=side; z+=step) {
            results[tid] += (int) world.is_monolith(x, z) * (step*step);
        }
    }
}

/* -------------------------------------------------------------------------- */

enum Variant {
    WORLD_PER_THREAD,
    WORLD_PER_BLOCK,
};

int main() {

    /* ------------------------------ */

    // Disable spin-wait on device syncronization
    hipSetDeviceFlags(hipDeviceScheduleBlockingSync);

    /* ------------------------------ */

    Variant variant;
    variant = WORLD_PER_THREAD;
    // variant = WORLD_PER_BLOCK;

    int start = 0;
    int seeds = 10000000;
    float* d_results;
    hipMalloc(&d_results, seeds * sizeof(float));

    /* ------------------------------ */

    if (variant == WORLD_PER_THREAD) {
        int thread = 32; // Ephemeral threads
        int blocks = (seeds + thread - 1) / thread;
        get_monoliths_world_per_thread<<<blocks, thread>>>(start, seeds, d_results);
    } else if (variant == WORLD_PER_BLOCK) {
        int thread = 64; // Full warp
        int blocks = seeds;
        get_monoliths_world_per_block<<<blocks, thread>>>(start, seeds, d_results);
    }

    /* ------------------------------ */

    hipDeviceSynchronize();

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
        return -1;
    }

    // Fixme: Make a iterative loop searching chunks of N seeds
    float* results = new float[seeds];
    hipMemcpy(results, d_results, seeds * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_results);

    /* ------------------------------ */
    // Print findings

    for (int i=0; i<seeds; i++) {
        if (results[i] > 0) {
            printf("Seed %llu area: %f\n", start + i, results[i]);
        }
    }

    return 0;
}